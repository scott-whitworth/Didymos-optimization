#include "hip/hip_runtime.h"
// Didymos Optimization Project using CUDA and a genetic algorithm

#include "../Earth_calculations/earthInfo.h"
#include "../Runge_Kutta/runge_kuttaCUDA.cuh" //for testing rk4simple
#include "../Config_Constants/config.h"
#include "../output.h"

#include <iostream> // cout
#include <iomanip>  // used for setw(), sets spaces between values output
#include <time.h>   // for seeding the random number generator
#include <random>
#include <chrono>

#define SECONDS_IN_YEAR 365.25*24*3600

// Used to see if the best individual is changing
// Returns true if the currentBest is not equal to previousBest
bool changeInBest(double previousBestPos, double previousBestVel, Individual currentBest, double dRate) {
    if (trunc(previousBestPos/dRate) != trunc(currentBest.posDiff/dRate)) {
        return true;
    }
    else {
        if (trunc(previousBestVel/dRate) != trunc(currentBest.velDiff/dRate)) {
            return true;
        }
        else return false;
    }
}

// Utility function to display the currently best individual onto the terminal while the algorithm is still running
// Input: Individual to be displayed (assumed to be the best individual of the pool) and the value for the current generation iterated
void terminalDisplay(Individual& individual, unsigned int currentGeneration) {
    std::cout << "\nGeneration: " << currentGeneration << std::endl;
    std::cout << "Best individual:" << std::endl;
    std::cout << "\tposDiff: " << individual.posDiff << std::endl;
    std::cout << "\tvelDiff: " << individual.velDiff << std::endl;
    std::cout << "\tcost: "    << individual.cost << std::endl;
}

// Assumes pool is sorted array of Individuals, used in determining if the loop continues
// Output: Returns true if top ten individuals within the pool are within the tolerance
bool allWithinTolerance(double tolerance, Individual * pool, unsigned int currentGeneration, const cudaConstants* cConstants) {
    // Uses for loop to pinpoint which individual is not in tolerance and display it to the terminal
    for (int i = 0; i < cConstants->best_count; i++) {
        if(pool[i].posDiff >= cConstants->pos_threshold) {  // This isn't ideal, Change to getCost once getCost gets fleshed out //if (pool[i].getCost() >= tolerance ) {
            return false;
        }
    }
    // If iterated through and all were within tolerance, success
    return true;
}

// The function that starts up and runs the genetic algorithm with a continous loop until the critera is met (number of individuals equal to best_count is below the threshold value)
double optimize(const int numThreads, const int blockThreads, const cudaConstants* cConstants, thruster<double> thrust) {
    double calcPerS = 0;

    time_t timeSeed = cConstants->time_seed;
    std::mt19937_64 rng(timeSeed);
    std::cout << "----------------------------------------------------------------------------------------------------" << std::endl;
    
    //sets up mutate file
    setMutateFile(cConstants);
    
    // input parameters for rk4Simple which are the same for each thread
    double timeInitial = 0; // the starting time of the trip is always defined as zero   
    double absTol = cConstants->rk_tol; // the tolerance is a constant number that is shared amongst all runs
    double stepSize = (orbitalPeriod - timeInitial) / cConstants->max_numsteps; // the starting step size- same for each run- note that the current step size varies throughout each run

    double currentAnneal = cConstants->anneal_initial;

    Individual *inputParameters = new Individual[numThreads]; // contains all input parameters besides those which are always common amongst every thread

    double previousBestPos = 0; // set to zero to ensure there is a difference between previousBest and currentBest on generation zero (see changeInBest function)
    double previousBestVel = 0;

    if (cConstants->random_start) {
        // Sets inputParameters to hold parameters that are randomly generated within a reasonable range
        for (int i = 0; i < numThreads; i++) { 
            double tripTime = SECONDS_IN_YEAR*(rng() % 10001 / 10000.0 + 1.0); // (1 <-> 2 years) * SECONDS_IN_YEAR
            double alpha = M_PI * 2*((static_cast<double>(rng()) / rng.max()) - 0.5); // -PI <-> PI
            double beta  = M_PI * ((static_cast<double>(rng()) / rng.max())); // 0 <-> PI
            double zeta  = M_PI * ((static_cast<double>(rng()) / rng.max()) - 0.5); // -PI/2 <-> PI/2

            coefficients<double> testcoeff;
            for (int j = 0; j < testcoeff.gammaSize; j++) {
                testcoeff.gamma[j] = rng() % 201/10.0 - 10.0; // -10.0 <-> 10.0
            }
            for (int j = 0; j < testcoeff.tauSize; j++) {
                testcoeff.tau[j] = rng() % 201/10.0 - 10.0; // -10.0 <-> 10.0
            }
            for (int j = 0; j < testcoeff.coastSize; j++) {
                testcoeff.coast[j] = rng() % 201/10.0 - 10.0; // -10.0 <-> 10.0
            }
        
            rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 
        
            inputParameters[i].startParams = example;
        }
    }
    // If not a random start, read from file using cConstants initial_start_file_address to get path
    else {
        // Sets inputParameters to hold initial individuals based from file optimizedVector.bin

        const int numStarts = 14; // the number of different sets of starting parameters in the input file

        std::ifstream starts;
        starts.open(cConstants->initial_start_file_address, std::ifstream::in|std::ios::binary); // a file containing the final parameters of converged results from CPU calculations        

        // sort the data into 2 dimensions
        // one row is one set of starting parameters
        // each column is a specific variable:
        double startDoubles;
        // arrayCPU needs to be updated to handle the fact that OPTIM_VARS now is defined from cConstants
        double arrayCPU[numStarts][OPTIM_VARS];
        
        for (int i = 0; i < OPTIM_VARS; i++) { // rows
            for (int j = 0; j < numStarts; j++) { // columns
                starts.read( reinterpret_cast<char*>( &startDoubles ), sizeof startDoubles );
                arrayCPU[j][i] = startDoubles;
            }
        }
        starts.close();

         // set every thread's input parameters to a set of final values from CPU calculations for use as a good starting point
        for (int i = 0; i < numThreads; i++) {
            int row = rng() % numStarts; // Choose a random row to get the parameters from

            double tripTime = arrayCPU[row][TRIPTIME_OFFSET];
            double alpha = arrayCPU[row][ALPHA_OFFSET];
            double beta = arrayCPU[row][BETA_OFFSET];
            double zeta = arrayCPU[row][ZETA_OFFSET];

            coefficients<double> testcoeff;
            for (int j = 0; j < testcoeff.gammaSize; j++) {
                testcoeff.gamma[j] = arrayCPU[row][j + GAMMA_OFFSET];
            }

            for (int j = 0; j < testcoeff.tauSize; j++) {
                testcoeff.tau[j] =  arrayCPU[row][j + TAU_OFFSET];
            }

            for (int j = 0; j < testcoeff.coastSize; j++) {
                testcoeff.coast[j] = arrayCPU[row][j + COAST_OFFSET];
            }

            rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 

            inputParameters[i].startParams = example;
        }
    }


    Individual *survivors = new Individual[cConstants->survivor_count]; // stores the winners of the head-to-head competition
    int newInd = numThreads; // the whole population is new the first time through the loop

    // Initialize the recording files if in record mode
    if (cConstants->record_mode == true) {
        initializeRecord(cConstants);
    }

    double generation = 0;    // A counter for number of generations calculated
    
    // A do-while loop that continues until it is determined that the pool of inputParameters has reached desired tolerance level for enough individuals (best_count)
    
    double currentDistance; // Contains value for how far away the best individual is from the tolerance value
    double tolerance = cConstants->pos_threshold; // Tolerance for what is an acceptable solution (currently just the position threshold which is furthest distance from the target allowed)
                                                  // This could eventually take into account velocity too and become a more complex calculation
    double dRate = 1.0e-8;

    do { // Set as a do while loop so that the algorithm is set to run atleast once
        // initialize positions for the new individuals starting at the index of the first new one and going to the end of the array
        initializePosition(inputParameters + (numThreads - newInd), newInd, cConstants);
        callRK(newInd, blockThreads, inputParameters + (numThreads - newInd), timeInitial, stepSize, absTol, calcPerS, thrust, cConstants); // calculate trajectories for new individuals

        // if we got bad results reset the Individual to random starting values (it may still be used for crossover) and set the final position to be way off so it gets replaced by a new Individual
        for (int k = 0; k < numThreads; k++) {
            if (isnan(inputParameters[k].finalPos.r) || isnan(inputParameters[k].finalPos.theta) || isnan(inputParameters[k].finalPos.z) 
                 || isnan(inputParameters[k].finalPos.vr) || isnan(inputParameters[k].finalPos.vtheta) || isnan(inputParameters[k].finalPos.vz)){
                
                std::cout << std::endl << std::endl << "NAN FOUND" << std::endl << std::endl;

                double tripTime = SECONDS_IN_YEAR*(std::rand() % 10001 / 10000.0 + 1.0);
                double alpha = M_PI * 2*((static_cast<double>(rng()) / rng.max()) - 0.5); // -PI <-> PI
                double beta  = M_PI *   ((static_cast<double>(rng()) / rng.max())); // 0 <-> PI
                double zeta  = M_PI *   ((static_cast<double>(rng()) / rng.max()) - 0.5); // -PI/2 <-> PI/2

                coefficients<double> testcoeff;
                if (thrust.type) {
                    for (int j = 0; j < testcoeff.gammaSize; j++) {
                        testcoeff.gamma[j] = rng() % 201/10.0 - 10.0;
                    }
                    for (int j = 0; j < testcoeff.tauSize; j++) {
                        testcoeff.tau[j] = rng() % 201/10.0 - 10.0;
                    }
                    for (int j = 0; j < testcoeff.coastSize; j++) {
                        testcoeff.coast[j] = rng() % 201/10.0 - 10.0;
                    }
                }
            
                rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 
        
                inputParameters[k].startParams = example;

                // Set to be a bad individual
                inputParameters[k].posDiff = 1.0;
                inputParameters[k].velDiff = 0.0;
             }

            // calculate its new cost function
            inputParameters[k].getCost(cConstants);
        }

        // Note to future development, should shuffle and sort be within selectWinners method?
        std::shuffle(inputParameters, inputParameters + numThreads, rng); // shuffle the Individiuals to use random members for the competition
        selectSurvivors(inputParameters, cConstants->survivor_count, survivors); // Choose which individuals are in survivors, not necessarrily only the best ones
        std::sort(inputParameters, inputParameters + numThreads); // put the individuals in order so we can replace the worst ones

        // Display a '.' to the terminal to show that a generation has been performed
        // This also serves to visually seperate the generation display on the terminal screen
        std::cout << '.';

        // Calculate how far the pool is from the ideal cost value (currently is the positionalDifference of the best individual)
        currentDistance = inputParameters[0].posDiff; // Change this later to take into account more than just the best individual and its position difference

        double new_anneal = currentAnneal * (1 - tolerance / currentDistance);
        
        Individual currentBest;
        if (static_cast<int>(generation) % cConstants->change_check == 0) { // Compare current best individual to that from CHANGE_CHECK many generations ago. If they are the same, change size of mutations
            currentBest = inputParameters[0];
          
            if ( !(changeInBest(previousBestPos, previousBestVel, currentBest, dRate)) ) { // previousBest starts at 0 to ensure changeInBest = true on generation 0
                currentAnneal = currentAnneal * cConstants->anneal_factor;
                std::cout << "\nnew anneal: " << currentAnneal << std::endl;
                if(trunc(inputParameters[0].posDiff/dRate)==0) { dRate = dRate/10; }
            }
            previousBestPos = currentBest.posDiff;
            previousBestVel = currentBest.velDiff;
        }

        // If in recording mode and write_freq reached, call the record method
        if (static_cast<int>(generation) % cConstants->write_freq == 0 && cConstants->record_mode == true) {
            recordGenerationPerformance(cConstants, inputParameters, generation, new_anneal, numThreads);
        }

        // Only call terminalDisplay every DISP_FREQ, not every single generation
        if ( static_cast<int>(generation) % cConstants->disp_freq == 0) {
            terminalDisplay(inputParameters[0], generation);
        }

        // Create a new generation and increment the generation counter
        newInd = newGeneration(survivors, inputParameters, cConstants->survivor_count, numThreads, new_anneal, cConstants, thrust, rng, generation);
        ++generation;
        
        // If the current distance is still higher than the tolerance we find acceptable, perform the loop again
    } while ( !allWithinTolerance(tolerance, inputParameters, generation, cConstants) );

    
    // output the best Individuals of the final generation, using writeTrajectoryToFile()
    // Files outputted allows plotting of solutions in matlab
    double *start = new double[OPTIM_VARS];

    // Write the final best and worst performing individuals to their respective files    
    recordGenerationPerformance(cConstants, inputParameters, generation, 0, numThreads);
    
    // std::ofstream progressiveOutput;
    // progressiveOutput.open("progressiveAnalysis.csv", std::ios::app);
    // progressiveOutput << std::endl << "seed:," << cConstants->time_seed << ",  ,generations:," << static_cast<int>(generation) << std::endl;
    // progressiveOutput << "rank,posDiff (au),velDiff (au/s),tripTime (s),alpha (rad),beta (rad),zeta (rad),";
    // if (thrust.type) {
    //     progressiveOutput << "gamma_a0,gamma_a1,gamma_b1,gamme_a2,gamme_b2,gamma_a3,gamma_b3,";
    //     progressiveOutput << "tau_a0,tau_a1,tau_b1,";
    //     progressiveOutput << "coast_a0,coast_a1,coast_b1,coast_a2,coast_b2,";
    // }
    // progressiveOutput << std::endl;
    // // Write the best individuals with best_count in total outputted in seperate binary files
    // for (int i = 0; i < cConstants->best_count; i++) {
    //     for (int j = 0; j < inputParameters[i].startParams.coeff.gammaSize; j++) {
    //         start[GAMMA_OFFSET + j] = inputParameters[i].startParams.coeff.gamma[j];
    //     }
    //     for (int j = 0; j < inputParameters[i].startParams.coeff.tauSize; j++) {
    //         start[TAU_OFFSET + j] = inputParameters[i].startParams.coeff.tau[j];
    //     }
    //     for (int j = 0; j < inputParameters[i].startParams.coeff.coastSize; j++) {
    //         start[COAST_OFFSET + j] = inputParameters[i].startParams.coeff.coast[j];
    //     }

    //     start[TRIPTIME_OFFSET] = inputParameters[i].startParams.tripTime;
    //     start[ALPHA_OFFSET] = inputParameters[i].startParams.alpha;
    //     start[BETA_OFFSET] = inputParameters[i].startParams.beta;
    //     start[ZETA_OFFSET] = inputParameters[i].startParams.zeta;

    //     // could instead use a ratio between position and velocity differnce as done in comparison of Individuals
    //     writeTrajectoryToFile(start, i+1, thrust, cConstants);
    //     progressiveAnalysis(progressiveOutput,i+1,inputParameters[i],cConstants);
    // }
    // progressiveOutput << std::endl;
    // progressiveOutput.close();

    // Only output the final best individual
    for (int j = 0; j < inputParameters[0].startParams.coeff.gammaSize; j++) {
        start[GAMMA_OFFSET + j] = inputParameters[0].startParams.coeff.gamma[j];
    }
    for (int j = 0; j < inputParameters[0].startParams.coeff.tauSize; j++) {
        start[TAU_OFFSET + j] = inputParameters[0].startParams.coeff.tau[j];
    }
    for (int j = 0; j < inputParameters[0].startParams.coeff.coastSize; j++) {
        start[COAST_OFFSET + j] = inputParameters[0].startParams.coeff.coast[j];
    }

    start[TRIPTIME_OFFSET] = inputParameters[0].startParams.tripTime;
    start[ALPHA_OFFSET] = inputParameters[0].startParams.alpha;
    start[BETA_OFFSET] = inputParameters[0].startParams.beta;
    start[ZETA_OFFSET] = inputParameters[0].startParams.zeta;

    // could instead use a ratio between position and velocity differnce as done in comparison of Individuals
    writeTrajectoryToFile(start, 1, thrust, cConstants);

    delete [] inputParameters;
    delete [] survivors;
    delete start;

    return calcPerS;
}

int main () {
    // display GPU properties and ensure we are using the right one
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "\n\nDevice Number: 0 \n";
    std::cout << "- Device name: " << prop.name << std::endl << std::endl;
    hipSetDevice(0);
    
    cudaConstants const * cConstants = new cudaConstants("../Config_Constants/genetic.config"); // Declare the genetic constants used, with file path being used
    // Display contents of cConstants resulting from reading the file onto the terminal
    std::cout << *cConstants << std::endl;

    // pre-calculate a table of Earth's position within possible mission time range
    //----------------------------------------------------------------
    // Define variables to be passed into EarthInfo that determines the range of time to be calculated, accessed from cConstants
    double startTime = cConstants->startTime;
    double endTime = cConstants->endTime; 
    double timeRes = cConstants->timeRes;

    launchCon = new EarthInfo(startTime, endTime, timeRes, cConstants); // a global variable to hold Earth's position over time

    // double timeStamp = startTime;
    
    // // File stream for outputting values that were calculated in EarthInfo constructor
    if (cConstants->record_mode) {
        std::ofstream earthValues;
        earthValues.open("EarthCheckValues.csv");
        // Set header row for the table to record values, with timeStamp
        earthValues << "TimeStamp, Radius, Theta, Z, vRadius, vTheta, vZ\n";
        while (timeStamp < endTime) {
            earthValues << timeStamp << "," << launchCon->getCondition(timeStamp);
            timeStamp += timeRes*24; // Increment to next day as timeRes is set to every hour
        }
        // Done recording earth calculations, close file and move on
        earthValues.close();
    }
    
    //----------------------------------------------------------------
    // Define the number of threads/individuals that will be used in optimize
    int blockThreads = cConstants->thread_block_size;
    int numThreads = cConstants->num_individuals;

    std::cout << std::endl << "running optimize() with " << blockThreads << " threads per block and " << numThreads << " total threads" << std::endl;

    thruster<double> thrust(cConstants);

    // Perform the optimization with optimize function
    optimize(numThreads, blockThreads, cConstants, thrust);

    // Now that the optimize function is done (assumed taht optimize() also records it), deallocate memory of the earth calculations and cudaConstants
    delete launchCon;
    delete cConstants;
    
    return 0;
}